#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h> 
#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}
template <unsigned int WarpSize>
__device__ __forceinline__ float warp_reduce(float sum)
{   
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); 
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}
template <const int ROW_PER_BLOCK>
__global__ void Sgemv_v2(float* d_mat,float* d_vector,float* d_res,int M,int N)
{
    const int bx = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int warpsize = 32;

    const int row = (bx*blockDim.y+ty)*ROW_PER_BLOCK;
    const int col = tx%32;

    const int cur_nums = warpsize/ROW_PER_BLOCK;
    const int cur_col = col%cur_nums;
    const int cur_row = row + col/cur_nums;

    if (cur_row<M)
    {
        float res =0;
        res += d_mat[cur_row*N+cur_col]*d_vector[cur_col];
        res = warp_reduce<cur_nums>(res);
        if (cur_col ==0) d_res[cur_row] = res;
    }
}
int main(int argc, char ** argv)
{
    if (argc!= 3)
    {
        printf("please input ./main M N");
        exit(0);
    }
    const int M = atoi(argv[1]);
    const int N = atoi(argv[2]);

    float * mat =(float*)malloc(M*N*sizeof(float));
    float * vector =(float*)malloc(N*sizeof(float));
    float * res =(float*)malloc(M*sizeof(float));
    float * res1 =(float*)malloc(M*sizeof(float));

    for (int i=0;i<M*N;i++)mat[i] = (float)i/13;
    for (int i=0; i<N;i++) vector[i] = (float)i/13;
    memset(res,0,M*sizeof(float));
    memset(res1,0,M*sizeof(float));

    float *d_mat;
    float *d_vector;
    float *d_res;
    const int ROW_PER_WARP = 2;
    

    hipMalloc(&d_mat,M*N*sizeof(float));
    hipMalloc(&d_vector,N*sizeof(float));
    hipMalloc(&d_res,M*sizeof(float));

    hipMemcpy(d_mat,mat,M*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_vector,vector,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_res,res,M*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid(M/8);
    dim3 Block(32,4);
    int nIter = 1000;
    hipEvent_t start ,stop;
    float ms = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int run = 0 ; run < nIter; run ++ ) {
        Sgemv_v2<ROW_PER_WARP><<< Grid, Block >>>(d_mat, d_vector, d_res, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    printf("ms : %.2f \n",ms/1000);
    hipMemcpy(res,d_res,M*sizeof(float),hipMemcpyDeviceToHost);



    // cublas
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    hipMemcpy(d_res, res1, M*sizeof(float), hipMemcpyHostToDevice);
    hipEvent_t start1 ,stop1;
    float ms1 = 0;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemv (blas_handle, HIPBLAS_OP_T, 
            N, M, &alpha, 
            d_mat, N, d_vector, 1, &beta, d_res, 1
        );
    }
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&ms1,start1,stop1);
    printf("ms : %.2f \n",ms1/1000);
    hipMemcpy(res1, d_res, M*sizeof(float), hipMemcpyDeviceToHost);
    hipblasDestroy(blas_handle); 
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M; i++) {
        double abs_err = fabs(res1[i] - res[i]);
        double dot_length = M;
        double abs_val = fabs(res[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, res[i], res1[i], eps);
            correct = false;
            break;
        }
    }
    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");

}
