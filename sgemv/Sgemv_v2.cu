#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h> 

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

// if N <= 16
template <
    const int ROW_PER_WARP
    > 
__global__ void Sgemv_v2( 
    float * __restrict__ A,
    float * __restrict__ x,
    float * __restrict__ y, 
    const int M,
    const int N) {
    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int warp_size=32;
    int laneId= tx % warp_size;
    int current_warp_row = (blockDim.y * bx + ty) * ROW_PER_WARP;
    const int kWarp_size = warp_size / ROW_PER_WARP;
    int kLaneId = laneId % kWarp_size;
    int current_thread_row = current_warp_row + laneId / kWarp_size;

    if(current_thread_row < M){
        float res=0;
        int current_col = kLaneId;
        res += A[current_thread_row * N + current_col] * x[current_col];
        res = warpReduceSum<kWarp_size>(res);
        if(kLaneId==0) y[current_thread_row]=res;
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("usage: ./main [M] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t N = atoi(argv[2]);

    size_t bytes_A = sizeof(float) * M * N;
    size_t bytes_x = sizeof(float) * N;
    size_t bytes_y = sizeof(float) * M;
    float* h_A = (float*)malloc(bytes_A);
    float* h_x = (float*)malloc(bytes_x);
    float* h_y = (float*)malloc(bytes_y);
    float* h_y1 = (float*)malloc(bytes_y);

    float* d_A;
    float* d_x;
    float* d_y;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_x, bytes_x));
    checkCudaErrors(hipMalloc(&d_y, bytes_y));

    const int WARP_SIZE=32;
    const int ROW_PER_WARP=2;
    const int THREAD_PER_BLOCK=128;
    const int WARP_PER_BLOCK=THREAD_PER_BLOCK/WARP_SIZE;
    const int ROW_PER_BLOCK=WARP_PER_BLOCK * ROW_PER_WARP;

    // 生成A的数据
    for( int i = 0; i < M * N; i++ ) {
        h_A[i] = (float)i/N;
    }

    // 生成x的数据
    for( int i = 0; i < N; i++ ) {
        h_x[i] = 1;
    }
    memset(h_y,0,M*sizeof(float));
    memset(h_y1,0,M*sizeof(float));

    int nIter = 1000;
    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_y, h_y, bytes_y, hipMemcpyHostToDevice));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimGrid(M/ROW_PER_BLOCK);
        dim3 dimBlock(32,THREAD_PER_BLOCK/WARP_SIZE);
        Sgemv_v2<ROW_PER_WARP><<< dimGrid, dimBlock >>>(d_A, d_x, d_y, M, N);
    }
    checkCudaErrors(hipMemcpy( h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    // cublas
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_y, h_y1, bytes_y, hipMemcpyHostToDevice));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemv (blas_handle, HIPBLAS_OP_T, 
            N, M, &alpha, 
            d_A, N, d_x, 1, &beta, d_y, 1
        );
    }
    checkCudaErrors(hipMemcpy( h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipblasDestroy(blas_handle); 
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M; i++) {
        double abs_err = fabs(h_y[i] - h_y1[i]);
        double dot_length = M;
        double abs_val = fabs(h_y[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_y[i], h_y1[i], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    
    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y1);
}
