#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256
#define FETCH_FLOAT2(pointer)(reinterpret_cast<float2*>(&(pointer))[0])
#define FETCH_FLOAT4(pointer)(reinterpret_cast<float4*>(&(pointer))[0])
__global__ void elementwise_add_2 (float* d_A,float *d_B,float *d_C)
{
    const int tid = (blockIdx.x*blockDim.x+threadIdx.x)*2;
    float2 reg_a = FETCH_FLOAT2(d_A[tid]);
    float2 reg_b = FETCH_FLOAT2(d_B[tid]);
    float2 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    FETCH_FLOAT2(d_C[tid]) = reg_c;
}
__global__ void elementwise_add(float* d_A,float *d_B,float *d_C)
{
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x)*4;
    float4 reg_a = FETCH_FLOAT4(d_A[tid]);
    float4 reg_b = FETCH_FLOAT4(d_B[tid]);
    float4 reg_c;
    reg_c.x = reg_a.x+reg_b.x;
    reg_c.y = reg_a.y+reg_b.y;
    reg_c.z = reg_a.z+reg_b.z;
    reg_c.w = reg_a.w+reg_b.w;
    FETCH_FLOAT4(d_C[tid]) = reg_c;   
}
bool check(float* a ,float * b ,int nums)
{
    for (int i=0;i<nums;i++)
    {
        if (a[i]!=b[i])return false;
    }
    return true;
}
int main()
{
    const int N=32*1024*1024;
    float * A =  (float*)malloc(N*sizeof(float));
    float * B =  (float*)malloc(N*sizeof(float));
    float * C =  (float*)malloc(N*sizeof(float));
    float * C1 =  (float*)malloc(N*sizeof(float));

    for (int i=0;i<N;i++)
    {
        A[i] = i/13;
        B[i] = i%13;
        C[i] = i/13 + i%13;
    }

    float * d_A ;
    float * d_B ;
    float * d_C ;

    hipMalloc(&d_A,N*sizeof(float));
    hipMalloc(&d_B,N*sizeof(float));
    hipMalloc(&d_C,N*sizeof(float));
    hipMemcpy(d_A,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,N*sizeof(float),hipMemcpyHostToDevice);
    hipEvent_t start ,stop;
    float ms = 0;
    dim3 Grid (N/THREAD_PER_BLOCK/4,1);
    dim3 Block(THREAD_PER_BLOCK,1);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i=0;i<1000;i++)
    {
        elementwise_add<<<Grid,Block>>>(d_A,d_B,d_C);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    printf("ms : %.2f \n",ms/1000);



    hipMemcpy(C1,d_C,N*sizeof(float),hipMemcpyDeviceToHost);
    if(check(C,C1,N))
    {
        printf("true \n");
    }
    else 
    {
        printf("wrong \n");
    }

    hipEvent_t start1 ,stop1;
    float ms1 = 0;
    dim3 Grid1 (N/THREAD_PER_BLOCK/2,1);
    dim3 Block1(THREAD_PER_BLOCK,1);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    for (int i=0;i<1000;i++)
    {
        elementwise_add_2<<<Grid1,Block1>>>(d_A,d_B,d_C);
    }
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&ms1,start1,stop1);
    printf("ms : %.2f \n",ms1/1000);


    hipMemcpy(C1,d_C,N*sizeof(float),hipMemcpyDeviceToHost);
    if(check(C,C1,N))
    {
        printf("true \n");
    }
    else 
    {
        printf("wrong \n");
    }    

    free(A);
    free(B);
    free(C);
    free(C1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}